#include <hip/hip_runtime.h>
#include <math.h>

#include <stdio.h>

__global__ void calcDominationMap(float *pFitnessArray, int numFitnessMeasures, int numGenomes, short *pDominationMap)
{
	int genomeIndex1 = blockIdx.x * blockDim.x + threadIdx.x;
	int genomeIndex2 = blockIdx.y * blockDim.y + threadIdx.y;

	if (genomeIndex1 >= numGenomes || genomeIndex2 >= numGenomes)
		return;
	
	int fitnessOffset1 = genomeIndex1*numFitnessMeasures;
	int fitnessOffset2 = genomeIndex2*numFitnessMeasures;

	int betterOrEqualCount = 0;
	int equalCount = 0;

	for (int i = 0 ; i < numFitnessMeasures ; i++)
	{
		float val1 = pFitnessArray[fitnessOffset1+i];
		float val2 = pFitnessArray[fitnessOffset2+i];

		if (val1 == val2)
		{
			equalCount++;
			betterOrEqualCount++;
		}
		else if (val1 < val2)
			betterOrEqualCount++;
	}

	int mapIndex = genomeIndex1*numGenomes+genomeIndex2;

	if (betterOrEqualCount == numFitnessMeasures && equalCount < numFitnessMeasures)
	{
		// 1 dominates 2
		pDominationMap[mapIndex] = 1;
	}
	else
		pDominationMap[mapIndex] = 0;
}

__global__ void calcDominatedByCount(int numGenomes, short *pDominationMap, int *pCount)
{
	int genomeIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (genomeIndex >= numGenomes)
		return;
	
	int offset = genomeIndex;
	int count = 0;

	for (int i = 0 ; i < numGenomes ; i++, offset += numGenomes)
	{
		if (pDominationMap[offset])
		{
			count++;
		}
	} 
	pCount[genomeIndex] = count; 
}

__global__ void calcDominatesList(int numGenomes, short *pDominationMap, int *pCount) // domination map will be reused for this
{
	int genomeIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (genomeIndex >= numGenomes)
		return;
	
	int offset = genomeIndex*numGenomes;
	int storeOffset = offset;
	int count = 0;

	for (int i = 0 ; i < numGenomes ; i++, offset++)
	{
		if (pDominationMap[offset])
		{
			pDominationMap[storeOffset] = i;
			storeOffset++;
			count++;
		}
	}

	pCount[genomeIndex] = count;
}

__global__ void parallelCopy(short *pDominationMap, int dstOffset, int srcOffset, int num)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= num)
		return;
	
	pDominationMap[dstOffset+i] = pDominationMap[srcOffset+i];
}

bool allocDeviceMemory(int numGenomes, int numFitnessMeasures, float **pDeviceFitnessArray, short **pDeviceDominationMap,
                       int **pDeviceDominatedByCount, int **pDeviceDominatesCount, short **pHostDominationMap)
{
	*pDeviceFitnessArray = 0;
	*pDeviceDominationMap = 0;
	*pDeviceDominatedByCount = 0;
	*pDeviceDominatesCount = 0;
	*pHostDominationMap = 0;

	if (hipMalloc((void **)pDeviceFitnessArray, numGenomes*numFitnessMeasures*sizeof(float)) != hipSuccess)
	{
		*pDeviceFitnessArray = 0;
		return false;
	}

	if (hipMalloc((void **)pDeviceDominationMap, numGenomes*numGenomes*sizeof(short)) != hipSuccess)
	{
		hipFree(*pDeviceFitnessArray);
		*pDeviceFitnessArray = 0;
		*pDeviceDominationMap = 0;
		return false;
	}

	if (hipMalloc((void **)pDeviceDominatedByCount, numGenomes*sizeof(int)) != hipSuccess)
	{
		hipFree(pDeviceFitnessArray);
		hipFree(pDeviceDominationMap);
		*pDeviceFitnessArray = 0;
		*pDeviceDominationMap = 0;
		*pDeviceDominatedByCount = 0;
		return false;
	}

	if (hipMalloc((void **)pDeviceDominatesCount, numGenomes*sizeof(int)) != hipSuccess)
	{
		hipFree(pDeviceFitnessArray);
		hipFree(pDeviceDominationMap);
		hipFree(pDeviceDominatedByCount);
		*pDeviceFitnessArray = 0;
		*pDeviceDominationMap = 0;
		*pDeviceDominatedByCount = 0;
		*pDeviceDominatesCount = 0;
		return false;
	}

	if (hipHostAlloc((void **)pHostDominationMap, numGenomes*numGenomes*sizeof(short), 0) != hipSuccess)
	{
		hipFree(pDeviceFitnessArray);
		hipFree(pDeviceDominationMap);
		hipFree(pDeviceDominatedByCount);
		hipFree(pDeviceDominatesCount);
		*pDeviceFitnessArray = 0;
		*pDeviceDominationMap = 0;
		*pDeviceDominatedByCount = 0;
		*pDeviceDominatesCount = 0;
		*pHostDominationMap = 0;
		return false;
	}

	return true;
}

void freeDeviceMemory(float *pDeviceFitnessArray, short *pDeviceDominationMap,
                      int *pDeviceDominatedByCount, int *pDeviceDominatesCount, short *pHostDominationMap)
{
	hipFree(pDeviceFitnessArray);
	hipFree(pDeviceDominationMap);
	hipFree(pDeviceDominatedByCount);
	hipFree(pDeviceDominatesCount);
	hipHostFree(pHostDominationMap);
}
	
void calcDominationInfo(int numGenomes, int numFitnessMeasures, float *pHostFitnessArray, short *pHostDominationMap,
			int *pHostDominatedByCount, int *pHostDominatesCount,
                        float *pDeviceFitnessArray, short *pDeviceDominationMap, 
			int *pDeviceDominatedByCount, int *pDeviceDominatesCount,
			int blockSize1, int blockSize2, int *pOffsets)
{
	hipMemcpy(pDeviceFitnessArray, pHostFitnessArray, numGenomes*numFitnessMeasures*sizeof(float), hipMemcpyHostToDevice);

	int blockSize = blockSize1;
	int numBlocks = numGenomes/blockSize;

	if ((numGenomes % blockSize) != 0)
		numBlocks++;

	dim3 threads(blockSize,blockSize);
	dim3 blocks(numBlocks, numBlocks);

	calcDominationMap<<<blocks, threads>>>(pDeviceFitnessArray, numFitnessMeasures, numGenomes, pDeviceDominationMap);

	blockSize = blockSize2;
	numBlocks = numGenomes/blockSize;

	if ((numGenomes % blockSize) != 0)
		numBlocks++;

	calcDominatedByCount<<<numBlocks, blockSize>>>(numGenomes, pDeviceDominationMap, pDeviceDominatedByCount);
	calcDominatesList<<<numBlocks, blockSize>>>(numGenomes, pDeviceDominationMap, pDeviceDominatesCount);

	hipMemcpy(pHostDominatedByCount, pDeviceDominatedByCount, numGenomes*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(pHostDominatesCount, pDeviceDominatesCount, numGenomes*sizeof(int), hipMemcpyDeviceToHost);
#if 0
	int destOffset = pHostDominatesCount[0];
	int roomLeft = numGenomes-pHostDominatesCount[0];
	int totalCount = pHostDominatesCount[0];
	
	pOffsets[0] = 0;

	for (int i = 1 ; i < numGenomes ; i++)
	{
		pOffsets[i] = totalCount;

		if (roomLeft == 0) // nothing to do then, already next to each other
		{
			destOffset += numGenomes;

			printf("HERE\n");
		}
		else
		{
			int srcOffset = i*numGenomes;
			int totalToCopy = pHostDominatesCount[i];

			while (totalToCopy > 0)
			{
				int numCopied = totalToCopy;

				if (numCopied > roomLeft)
					numCopied = roomLeft;

				// blockSize is still ok
				numBlocks = numCopied/blockSize;
				if ((numCopied % blockSize) != 0)
					numBlocks++;
				
				parallelCopy<<<numBlocks, blockSize>>>(pDeviceDominationMap, destOffset, srcOffset, numCopied);

				destOffset += numCopied;
				srcOffset += numCopied;
				totalToCopy -= numCopied;
			}
		}

		roomLeft += (numGenomes-pHostDominatesCount[i]);
		totalCount += pHostDominatesCount[i];
	}

	printf("%d vs %d\n",totalCount, numGenomes*numGenomes);
	cudaMemcpy(pHostDominationMap, pDeviceDominationMap, totalCount*sizeof(short), cudaMemcpyDeviceToHost);
#else
	for (int i = 0 ; i < numGenomes ; i++)
		pOffsets[i] = i*numGenomes;

	hipMemcpy(pHostDominationMap, pDeviceDominationMap, numGenomes*numGenomes*sizeof(short), hipMemcpyDeviceToHost);
#endif

}

bool checkCuda(int *pBlockSize1, int *pBlockSize2)
{
	int numDevices = 0;

	hipGetDeviceCount(&numDevices);

	if (numDevices == 0)
		return false;

	// TODO: For now, we only check device 0

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	if (deviceProp.major == 9999 && deviceProp.minor == 9999)
		return false;

	int maxThreads = deviceProp.maxThreadsPerBlock;

	*pBlockSize2 = maxThreads;

	double r = sqrt((double)maxThreads);

	*pBlockSize1 = (int)r;

	return true;
}

